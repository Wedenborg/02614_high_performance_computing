#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
*/

#ifdef _OPENMP
#include <omp.h>
#endif

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "alloc3d.h"
#include "print.h"

#ifdef _JACOBI
#include "jacobi.h"
#endif

#ifdef _JACOBIGPU1
#include "jacobi_gpu1.h"
#endif

#ifdef _JACOBIGPU2
#include "jacobi_gpu2.h"
#endif

#ifdef _JACOBIGPU3
#include "jacobi_gpu3.h"
#endif

#ifdef _JACOBIGPU4
#include "jacobi_gpu4.h"
#endif

//#define N 2880
#define N_DEFAULT 1000

int main(int argc, char *argv[]) {
    printf("%d \n", 1000000);
    int 	N = N_DEFAULT;
    int 	iter_max = 1000;
    double	tolerance;
    double	start_T;
    int     output_type =1;
    #ifdef _JACOBIGPU1
    char    *output_prefix ="Jacobi_gpu1.res";
    #endif

    #ifdef _JACOBIGPU2
    char    *output_prefix ="Jacobi_gpu2.res";
    #endif

    #ifdef _JACOBIGPU3
    char    *output_prefix ="Jacobi_gpu3.res";
    #endif

    
    #ifdef _JACOBIGPU4
    char    *output_prefix ="Jacobi_gpu4.res";
    #endif

    char    *output_ext = "";
    char    output_filename[FILENAME_MAX];

    double *** h_u = NULL;
    double *** h_v = NULL;
    double *** h_f = NULL;
    double *** d_u = NULL;
    double *** d_v = NULL;
    double *** d_f = NULL;

    int i,j,k; // indicies

    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    N = N+2;
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    tolerance = atof(argv[3]);  // tolerance
    start_T   = atof(argv[4]);  // start T for all inner grid points
    if (argc==6){
        output_type = atoi(argv[5]);
    }
    printf("%d \n",N);
    // speed up test 
    double ts, te;
    
    // allocate memory in CPU
    if ( (h_u = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array h_u: allocation failed");
        exit(-1);
    }
    // Initialize first guess as zero. 
    if ( (h_v = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array h_v: allocation failed");
        exit(-1);
    }
    // Allocating a f matrix
    if ( (h_f = d_malloc_3d(N, N, N)) == NULL ) {
    perror("array h_f: allocation failed");
    exit(-1);
    }

    // allocate memory in GPU
    if ( (d_u = d_malloc_3d_gpu(N, N, N)) == NULL ) {
            perror("array d_u: allocation failed");
            exit(-1);
    }
    
        // Initialize first guess as zero. 
    if ( (d_v = d_malloc_3d_gpu(N, N, N)) == NULL ) {
            perror("array d_v: allocation failed");
            exit(-1);
    }
    
        // Allocating a f matrix
    if ( (d_f = d_malloc_3d_gpu(N, N, N)) == NULL ) {
        perror("array d_f: allocation failed");
        exit(-1);
    }
    
    // init u, f
    #pragma omp parallel for default(none) shared(h_u,h_v,h_f, N, start_T) private( i, j, k)
    for( i =0; i < N; i++){
        for( j = 0; j < N; j++){
            for( k = 0; k < N; k++){
                if( i==0 || i==N-1 || j == N-1|| k == 0 || k == N-1  ){
                    h_u[i][j][k] = 20;
                    h_v[i][j][k] = 20;
                } else if (j == 0){
                    h_u[i][j][k] = 0;
                    h_v[i][j][k] = 0;
                } else {
                    h_u[i][j][k] = start_T ;
                    h_v[i][j][k] = start_T ;
                }
                if( i >= 0 && i <= 5./8.*N*0.5 - 1 && j >= 0 && j <= .5*N*0.5-1 && k >= 1./3.*0.5*N-1  && k <= N*0.5-1 ){
                    h_f[i][j][k] = 200;
                } else{
                    h_f[i][j][k] = 0;
                }
            }
        }
    }
    // slut omp

    for( int i =0; i < N; i++){
         for( int j = 0; j < N; j++){
             for( int k = 0; k < N; k++){
                 if( i >= 0 && i <= 5./8.*N*0.5 - 1 && j >= 0 && j <= .5*N*0.5-1 && k >= 1./3.*0.5*N-1  && k <= N*0.5-1 ){
                     h_f[i][j][k] = 200;
                 } else{
                     h_f[i][j][k] = 0;
                }
             }
         }
     }
    
    // Transfer matrix to device
    transfer_3d(d_u, h_u, N, N, N , hipMemcpyHostToDevice);
    transfer_3d(d_v, h_v, N, N, N , hipMemcpyHostToDevice);
    transfer_3d(d_f, h_f, N, N, N , hipMemcpyHostToDevice);   

    #ifdef _JACOBIGPU1

    ts = omp_get_wtime();
    jacobi_serial<<<1, 1>>>(N, d_u, d_v, d_f, iter_max);
    te = omp_get_wtime() - ts;
    printf("%lf \n",te);

    #endif

    #ifdef _JACOBIGPU2

    #define K 10
    
    dim3 dimGrid(ceil((double)N/K),ceil((double)N/K),ceil((double)N/K));
    dim3 dimBlock(K,K,K);
    ts = omp_get_wtime();
    int counter = 0;
    do{   
        double*** tmp = d_u;
        d_u = d_v;
        d_v = tmp;

    // printf("%d",counter);    
    jacobi_per_elem<<<dimGrid,dimBlock>>>(N, d_u, d_v, d_f, iter_max);
    checkCudaErrors(hipDeviceSynchronize());
    counter++;
    } while (counter <iter_max);

    te = omp_get_wtime() - ts;
    //printf("%lf \n",te);

    #endif 


    
    #ifdef _JACOBIGPU3
    double ***d0_u,***d1_u, ***d0_v,***d1_v,***d0_f,***d1_f;

    //hipMalloc((void*) &d0_A, (N*N*N)/2);
    //hipMalloc((void*) &d1_A, (N*N*N)/2);
    if ( (d0_u = d_malloc_3d_gpu(N, N, N/2)) == NULL ) {
        perror("array d_u: allocation failed");
        exit(-1);
    }
    if ( (d1_u = d_malloc_3d_gpu(N, N, N/2)) == NULL ) {
        perror("array d_u: allocation failed");
        exit(-1);
    }
    transfer_3d_from_1d(d0_u, h_u[0][0], N, N, N/2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(d1_u, h_u[0][0] + N*N*N/2, N, N, N/2 , hipMemcpyHostToDevice);

    if ( (d0_v = d_malloc_3d_gpu(N, N, N/2)) == NULL ) {
        perror("array d_u: allocation failed");
        exit(-1);
    }
    if ( (d1_v = d_malloc_3d_gpu(N, N, N/2)) == NULL ) {
        perror("array d_u: allocation failed");
        exit(-1);
    }
    transfer_3d_from_1d(d0_v, h_u[0][0], N, N, N/2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(d1_v, h_u[0][0]+N*N*N/2, N, N, N/2 , hipMemcpyHostToDevice);

    if ( (d0_f = d_malloc_3d_gpu(N, N, N/2)) == NULL ) {
        perror("array d_u: allocation failed");
        exit(-1);
    }
    if ( (d1_f = d_malloc_3d_gpu(N, N, N/2)) == NULL ) {
        perror("array d_u: allocation failed");
        exit(-1);
    }
    transfer_3d_from_1d(d0_f, h_u[0][0], N, N, N/2, hipMemcpyHostToDevice);
    transfer_3d_from_1d(d1_f, h_u[0][0] + N*N*N/2, N, N, N/2 , hipMemcpyHostToDevice);

    printf("i=%i j=%i k=%i | u=%f v=%f f=%f\n", i, j, k, h_u[10][10][10], h_v[10][10][10], f[10][10][10]);
    }

    ts = omp_get_wtime();


    #define K 10
    
    dim3 dimGrid(ceil((double)N/(K*2)),ceil((double)N/(K*2)),ceil((double)N/(K*2)));
    dim3 dimBlock(K,K,K);
    ts = omp_get_wtime();
    int counter = 0;
    do{   
        double*** tmp0 = d0_u;
        d0_u = d0_v;
        d0_v = tmp0;   

        double*** tmp1 = d1_u;
        d1_u = d1_v;
        d1_v = tmp1; 

        hipSetDevice(0);
        hipDeviceEnablePeerAccess(1,0);
        jacobi_dual1<<<dimGrid,dimBlock>>>(N, d0_u, d0_v, d0_f, d1_v,iter_max);
        

        hipSetDevice(1);
        hipDeviceEnablePeerAccess(0,0);
        jacobi_dual2<<<dimGrid,dimBlock>>>(N, d1_u, d1_v, d1_f, d0_v, iter_max);
        checkCudaErrors(hipDeviceSynchronize());

        hipSetDevice(0);
        checkCudaErrors(hipDeviceSynchronize());
      
    counter++;
    printf("%d \n",counter);
    } while (counter <iter_max);

    transfer_3d_to_1d(h_u[0][0],d0_u, N, N, N/2, hipMemcpyDeviceToHost);
    transfer_3d_to_1d(h_u[0][0]+ N*N*N/2,d1_u, N, N, N/2 , hipMemcpyDeviceToHost);
    transfer_3d_to_1d(h_v[0][0],d0_v, N, N, N/2, hipMemcpyDeviceToHost);
    transfer_3d_to_1d(h_v[0][0]+ N*N*N/2,d1_v, N, N, N/2 , hipMemcpyDeviceToHost);
    transfer_3d_to_1d(h_f[0][0],d0_f, N, N, N/2, hipMemcpyDeviceToHost);
    transfer_3d_to_1d(h_f[0][0]+ N*N*N/2,d1_f, N, N, N/2 , hipMemcpyDeviceToHost);

    te = omp_get_wtime() - ts;
    te = omp_get_wtime() - ts;
    printf("%lf \n",te);

    #endif
    

    #ifdef _JACOBIGPU4
    double* res_d = NULL, res_h;
    //res = 0;
    hipMalloc((void**)&res_d, sizeof(double));
    #define K 10
    
    dim3 dimGrid(ceil((double)N/K),ceil((double)N/K),ceil((double)N/K));
    dim3 dimBlock(K,K,K);
    ts = omp_get_wtime();
    int counter = 0;
    do{   
        double*** tmp = d_u;
        d_u = d_v;
        d_v = tmp;

    // printf("%d",counter);
    hipMemset(res_d, 0, sizeof(double));    
    jacobi_stopTest<<<dimGrid,dimBlock>>>(N, d_u, d_v, d_f, iter_max,res_d);
    hipMemcpy(&res_h, res_d, sizeof(double), hipMemcpyDeviceToHost);
    //printf("%lf",*res);
    
    checkCudaErrors(hipDeviceSynchronize());
    counter++;
    
    } while (counter <iter_max || tolerance< res_h);

    te = omp_get_wtime() - ts;
    //printf("%lf \n",te);

    #endif 
    
    // Transfer result to host
    transfer_3d(h_u, d_u, N, N, N , hipMemcpyDeviceToHost);
    transfer_3d(h_v, d_v, N, N, N , hipMemcpyDeviceToHost);
    transfer_3d(h_f, d_f, N, N, N , hipMemcpyDeviceToHost); 

    // de-allocate memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_f);

    switch(output_type){
        case 0:
        // no output 
        break;

        case 1:
        output_ext=".vtk";
        sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "Write VTK file to %s: ", output_filename);
	    print_vtk(output_filename, N, h_u);
	    break;
    }

    free(h_u);
    free(h_v);
    free(h_f);

    return(0);
}
