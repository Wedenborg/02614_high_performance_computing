/* jacobi.c - Poisson problem in 3d
 * 
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#ifdef _OPENMP
#include <omp.h>
#endif

__host__ __device__ 
void jacobi(int N, double ***u, double ***v, double ***f, int iter_max) {
    //double stopTest = 100000;
    int counter =0;
    int i,j,k;

    //while(stopTest>tolerance && counter < iter_max){
    //    stopTest =0.0;
    do{
        #pragma omp parallel default(none) shared(u, v, f, N) private(i, j, k) 
        {
        #pragma omp for
        for( i =0; i < N; i++){
            for( j = 0; j < N; j++){
                for( k = 0; k < N; k++){
                    v[i][j][k] = u[i][j][k];
                }
            }
        }

        // #pragma omp for reduction(+: stopTest)
        for( i =1; i < N-1; i++){
            for( j = 1; j < N-1; j++){
                for( k = 1; k < N-1; k++){
	                u[i][j][k] = 1./6.*(v[i-1][j][k]+v[i+1][j][k]+v[i][j-1][k]+v[i][j+1][k]+v[i][j][k-1]+v[i][j][k+1] + 1./((N-2)*(N-2)) * f[i][j][k]); //formula and matrix      

                    // stopTest +=(u[i][j][k]-v[i][j][k])*(u[i][j][k]-v[i][j][k]);
                    
	            }
            }
        }
        } //End Parallel
        counter++;
        
    //}
    } while (counter <iter_max);
}

// Kernel to be launched on a single thread
__global__
void jacobi_serial(int N, double ***u, double ***v, double ***f, int iter_max)
{
    jacobi(N, u, v, f, iter_max);
} 


