#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */
#ifdef _OPENMP
#include <omp.h>
#endif

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "alloc3d.h"
#include "print.h"

#ifdef _JACOBI
#include "jacobi.h"
#endif

#ifdef _JACOBIGPU1
#include "jacobi_gpu1.h"
#endif

#ifdef _JACOBIGPU2
#include "jacobi_gpu2.h"
#endif

//#define N 2880
#define N_DEFAULT 1000

int main(int argc, char *argv[]) {
    printf("%d \n", 1000000);
    int 	N = N_DEFAULT;
    int 	iter_max = 1000;
    double	tolerance;
    double	start_T;
    int     output_type =1;
    #ifdef _JACOBIGPU1
    char    *output_prefix ="Jacobi_gpu1.res";
    #endif

    #ifdef _JACOBIGPU2
    char    *output_prefix ="Jacobi_gpu1.res";
    #endif

    char    *output_ext = "";
    char    output_filename[FILENAME_MAX];


    double *** h_u = NULL;
    double *** h_v = NULL;
    double *** h_f = NULL;
    double *** d_u = NULL;
    double *** d_v = NULL;
    double *** d_f = NULL;


    int i,j,k; // indicies

    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    N = N+2;
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    tolerance = atof(argv[3]);  // tolerance
    start_T   = atof(argv[4]);  // start T for all inner grid points
    if (argc==6){
        output_type = atoi(argv[5]);
    }
    printf("%d \n",N);
    // speed up test 
    double ts, te;
    
    // allocate memory in CPU
    if ( (h_u = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array h_u: allocation failed");
        exit(-1);
    }
    // Initialize first guess as zero. 
    if ( (h_v = d_malloc_3d(N, N, N)) == NULL ) {
        perror("array h_v: allocation failed");
        exit(-1);
    }
    // Allocating a f matrix
    if ( (h_f = d_malloc_3d(N, N, N)) == NULL ) {
    perror("array h_f: allocation failed");
    exit(-1);
    }
    

    // allocate memory in GPU
    if ( (d_u = d_malloc_3d_gpu(N, N, N)) == NULL ) {
            perror("array d_u: allocation failed");
            exit(-1);
    }
    
        // Initialize first guess as zero. 
    if ( (d_v = d_malloc_3d_gpu(N, N, N)) == NULL ) {
            perror("array d_v: allocation failed");
            exit(-1);
    }
    
        // Allocating a f matrix
    if ( (d_f = d_malloc_3d_gpu(N, N, N)) == NULL ) {
        perror("array d_f: allocation failed");
        exit(-1);
    }
    
    // init u, f
    #pragma omp parallel for default(none) shared(h_u,h_f, N, start_T) private( i, j, k)
    for( i =0; i < N; i++){
        for( j = 0; j < N; j++){
            for( k = 0; k < N; k++){
                if( i==0 || i==N-1 || j == N-1|| k == 0 || k == N-1  ){
                    h_u[i][j][k] = 20;
                } else if (j == 0){
                    h_u[i][j][k] = 0;
                } else {
                    h_u[i][j][k] = start_T ;
                }
                if( i >= 0 && i <= 5./8.*N*0.5 - 1 && j >= 0 && j <= .5*N*0.5-1 && k >= 1./3.*0.5*N-1  && k <= N*0.5-1 ){
                    h_f[i][j][k] = 200;
                } else{
                    h_f[i][j][k] = 0;
                }
            }
        }
    }
    // slut omp

    for( int i =0; i < N; i++){
         for( int j = 0; j < N; j++){
             for( int k = 0; k < N; k++){
                 if( i >= 0 && i <= 5./8.*N*0.5 - 1 && j >= 0 && j <= .5*N*0.5-1 && k >= 1./3.*0.5*N-1  && k <= N*0.5-1 ){
                     h_f[i][j][k] = 200;
                 } else{
                     h_f[i][j][k] = 0;
                }
             }
         }
     }
    
    // Transfer matrix to device
    transfer_3d(d_u, h_u, N, N, N , hipMemcpyHostToDevice);
    transfer_3d(d_v, h_v, N, N, N , hipMemcpyHostToDevice);
    transfer_3d(d_f, h_f, N, N, N , hipMemcpyHostToDevice);   

    #ifdef _JACOBIGPU1

    ts = omp_get_wtime();
    jacobi_serial<<<1, 1>>>(N, d_u, d_v, d_f, iter_max);
    te = omp_get_wtime() - ts;
    printf("%lf \n",te);

    #endif


    #ifdef _JACOBIGPU2

    #define K 10
    
    dim3 dimGrid(ceil((double)N/K),ceil((double)N/K),ceil((double)N/K));
    dim3 dimBlock(K,K,K);
    ts = omp_get_wtime();
    int counter =0;
    do{
        
    jacobi_per_elem<<<dimGrid,dimBlock>>>(N-2, d_u, d_v, d_f, iter_max);
    //jacobi_per_elem<<<1,1>>>(N+2, d_u, d_v, d_f, iter_max);
    checkCudaErrors(hipDeviceSynchronize());
    counter++;

    } while (counter <iter_max);
    te = omp_get_wtime() - ts;
    printf("%lf \n",te);

    #endif 

    // checkCudaErrors(hipDeviceSynchronize());
    
    // Transfer result to host
    transfer_3d(h_u, d_u, N, N, N , hipMemcpyDeviceToHost);
    transfer_3d(h_v, d_v, N, N, N , hipMemcpyDeviceToHost);
    transfer_3d(h_f, d_f, N, N, N , hipMemcpyDeviceToHost); 

    // print h_U til .vyk burde virke... 



    // de-allocate memory
    //free_gpu(d_u);
    //free_gpu(d_v);
    //free_gpu(d_f);

    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_f);

    //hipHostFree(h_u);
    //hipHostFree(h_v);
    //hipHostFree(h_f);



    switch(output_type){
        case 0:
        // no output 
        break;

        case 1:
        output_ext=".vtk";
        sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "Write VTK file to %s: ", output_filename);
	    print_vtk(output_filename, N, h_u);
	    break;
    }
    free(h_u);
    free(h_v);
    free(h_f);

    return(0);
}
