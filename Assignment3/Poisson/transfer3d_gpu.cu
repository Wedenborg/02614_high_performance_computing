#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

void
transfer_3d(double ***dst, double ***src, int m, int n, int k, int flag)
{
    long nPtr = m + m * n;
    long nBlk = m * n * k;

    // we only transfer the value block
    checkCudaErrors( hipMemcpy((double *) dst + nPtr,
                                (double *) src + nPtr,
                                nBlk * sizeof(double),
                                (hipMemcpyKind) flag) );
}
